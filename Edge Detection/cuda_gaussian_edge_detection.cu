#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <cmath>
#include <vector>
#include <chrono>

#define CUDA_CHECK(call)                                                      \
    {                                                                         \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__      \
                      << " - " << hipGetErrorString(err) << std::endl;       \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    }

__constant__ float d_kernel[5][5];
#define BLOCK_SIZE 16
__global__ void GaussianBlurKernel(float* d_image, float* d_blurred, int width, int height) {
    // Shared memory for the tile
    __shared__ float tile[BLOCK_SIZE + 4][BLOCK_SIZE + 4];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    // Load shared memory with appropriate boundary handling
    int sharedX = tx + 2; // +2 for kernel offset
    int sharedY = ty + 2;
    if (x < width && y < height) {
        tile[sharedY][sharedX] = d_image[y * width + x];
    } else {
        tile[sharedY][sharedX] = 0.0f; // Zero-padding for out-of-bound pixels
    }

    // Load additional boundary pixels into shared memory
    if (tx < 2) {
        // Left border
        if (x >= 2) {
            tile[sharedY][tx] = d_image[y * width + (x - 2)];
        } else {
            tile[sharedY][tx] = 0.0f; // Padding
        }

        // Right border
        if (x + blockDim.x < width) {
            tile[sharedY][sharedX + blockDim.x] = d_image[y * width + (x + blockDim.x)];
        } else {
            tile[sharedY][sharedX + blockDim.x] = 0.0f;
        }
    }

    if (ty < 2) {
        // Top border
        if (y >= 2) {
            tile[ty][sharedX] = d_image[(y - 2) * width + x];
        } else {
            tile[ty][sharedX] = 0.0f;
        }

        // Bottom border
        if (y + blockDim.y < height) {
            tile[sharedY + blockDim.y][sharedX] = d_image[(y + blockDim.y) * width + x];
        } else {
            tile[sharedY + blockDim.y][sharedX] = 0.0f;
        }
    }

    // Wait for all threads to finish loading
    __syncthreads();

    // Perform the convolution only for valid pixels
    if (x < width && y < height) {
        float sum = 0.0f;
        for (int ky = 0; ky < 5; ++ky) {
            for (int kx = 0; kx < 5; ++kx) {
                sum += d_kernel[ky][kx] * tile[sharedY + ky - 2][sharedX + kx - 2];
            }
        }
        d_blurred[y * width + x] = sum;
    }
}


#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// Move Sobel kernels to constant memory
__constant__ int Gx[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int Gy[3][3] = {
    { 1,  2,  1},
    { 0,  0,  0},
    {-1, -2, -1}
};

__global__ void SobelKernel(float* d_blurred, float* d_gradient, float* d_direction, int width, int height) {
    // Calculate global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate indices in shared memory (+1 for halo)
    int s_x = threadIdx.x + 1;
    int s_y = threadIdx.y + 1;

    // Declare shared memory with halo regions
    __shared__ float s_data[BLOCK_SIZE_Y + 2][BLOCK_SIZE_X + 2];

    // Load central data into shared memory
    if (x < width && y < height)
        s_data[s_y][s_x] = d_blurred[y * width + x];
    else
        s_data[s_y][s_x] = 0.0f;

    // Load halo regions
    // Left and right halos
    if (threadIdx.x == 0) {
        int x_left = x - 1;
        s_data[s_y][s_x - 1] = (x_left >= 0 && y < height) ? d_blurred[y * width + x_left] : 0.0f;
    }
    if (threadIdx.x == blockDim.x - 1) {
        int x_right = x + 1;
        s_data[s_y][s_x + 1] = (x_right < width && y < height) ? d_blurred[y * width + x_right] : 0.0f;
    }

    // Top and bottom halos
    if (threadIdx.y == 0) {
        int y_top = y - 1;
        s_data[s_y - 1][s_x] = (x < width && y_top >= 0) ? d_blurred[y_top * width + x] : 0.0f;
    }
    if (threadIdx.y == blockDim.y - 1) {
        int y_bottom = y + 1;
        s_data[s_y + 1][s_x] = (x < width && y_bottom < height) ? d_blurred[y_bottom * width + x] : 0.0f;
    }

    // Corner halos
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int x_left = x - 1, y_top = y - 1;
        s_data[s_y - 1][s_x - 1] = (x_left >= 0 && y_top >= 0) ? d_blurred[y_top * width + x_left] : 0.0f;
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0) {
        int x_right = x + 1, y_top = y - 1;
        s_data[s_y - 1][s_x + 1] = (x_right < width && y_top >= 0) ? d_blurred[y_top * width + x_right] : 0.0f;
    }
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1) {
        int x_left = x - 1, y_bottom = y + 1;
        s_data[s_y + 1][s_x - 1] = (x_left >= 0 && y_bottom < height) ? d_blurred[y_bottom * width + x_left] : 0.0f;
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) {
        int x_right = x + 1, y_bottom = y + 1;
        s_data[s_y + 1][s_x + 1] = (x_right < width && y_bottom < height) ? d_blurred[y_bottom * width + x_right] : 0.0f;
    }

    // Synchronize to ensure all shared memory is loaded
    __syncthreads();

    // Perform convolution if within image bounds
    if (x < width && y < height) {
        float sumX = 0.0f, sumY = 0.0f;

        // Unrolled convolution
        sumX += -1 * s_data[s_y - 1][s_x - 1];
        sumX +=  0 * s_data[s_y - 1][s_x];
        sumX +=  1 * s_data[s_y - 1][s_x + 1];
        sumX += -2 * s_data[s_y][s_x - 1];
        sumX +=  0 * s_data[s_y][s_x];
        sumX +=  2 * s_data[s_y][s_x + 1];
        sumX += -1 * s_data[s_y + 1][s_x - 1];
        sumX +=  0 * s_data[s_y + 1][s_x];
        sumX +=  1 * s_data[s_y + 1][s_x + 1];

        sumY +=  1 * s_data[s_y - 1][s_x - 1];
        sumY +=  2 * s_data[s_y - 1][s_x];
        sumY +=  1 * s_data[s_y - 1][s_x + 1];
        sumY +=  0 * s_data[s_y][s_x - 1];
        sumY +=  0 * s_data[s_y][s_x];
        sumY +=  0 * s_data[s_y][s_x + 1];
        sumY += -1 * s_data[s_y + 1][s_x - 1];
        sumY += -2 * s_data[s_y + 1][s_x];
        sumY += -1 * s_data[s_y + 1][s_x + 1];

        // Compute gradient magnitude and direction
        d_gradient[y * width + x] = sqrtf(sumX * sumX + sumY * sumY);
        d_direction[y * width + x] = atan2f(sumY, sumX);
    }
}


__global__ void NonMaxSuppressionKernel(float* d_gradient, float* d_direction, float* d_edges, int width, int height) {
    // Implement non-maximum suppression logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    float direction = d_direction[y * width + x];
    float magnitude = d_gradient[y * width + x];
    int dx1 = 0, dy1 = 0, dx2 = 0, dy2 = 0;

    // Map direction to nearest 0, 45, 90, or 135 degrees
    float angle = fmodf(direction + M_PI, M_PI) * (180.0f / M_PI);
    if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle <= 180)) {
        dx1 = 1; dy1 = 0; dx2 = -1; dy2 = 0;
    } else if (angle >= 22.5 && angle < 67.5) {
        dx1 = 1; dy1 = 1; dx2 = -1; dy2 = -1;
    } else if (angle >= 67.5 && angle < 112.5) {
        dx1 = 0; dy1 = 1; dx2 = 0; dy2 = -1;
    } else if (angle >= 112.5 && angle < 157.5) {
        dx1 = -1; dy1 = 1; dx2 = 1; dy2 = -1;
    }

    int neighbor1X = min(max(x + dx1, 0), width - 1);
    int neighbor1Y = min(max(y + dy1, 0), height - 1);
    int neighbor2X = min(max(x + dx2, 0), width - 1);
    int neighbor2Y = min(max(y + dy2, 0), height - 1);

    float neighbor1 = d_gradient[neighbor1Y * width + neighbor1X];
    float neighbor2 = d_gradient[neighbor2Y * width + neighbor2X];

    if (magnitude >= neighbor1 && magnitude >= neighbor2) {
        d_edges[y * width + x] = magnitude;
    } else {
        d_edges[y * width + x] = 0.0f;
    }
}

__global__ void DoubleThresholdKernel(float* d_edges, float highThreshold, float lowThreshold, int width, int height) {
    // Implement double thresholding logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float value = d_edges[y * width + x];
    if (value >= highThreshold) {
        d_edges[y * width + x] = 1.0f; // Strong edge
    } else if (value >= lowThreshold) {
        d_edges[y * width + x] = 0.5f; // Weak edge
    } else {
        d_edges[y * width + x] = 0.0f; // Non-edge
    }
}

__global__ void HysteresisKernel(float* d_edges, int width, int height) {
    // Implement edge tracking by hysteresis
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    if (d_edges[y * width + x] != 0.5f) return; // Process only weak edges

    bool connectedToStrong = false;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;

            int neighborX = min(max(x + dx, 0), width - 1);
            int neighborY = min(max(y + dy, 0), height - 1);
            if (d_edges[neighborY * width + neighborX] == 1.0f) {
                connectedToStrong = true;
                break;
            }
        }
        if (connectedToStrong) break;
    }

    d_edges[y * width + x] = connectedToStrong ? 1.0f : 0.0f;
}

// Host Function Declarations
void processSingleScale(float* d_image, float* d_output, int width, int height);
void combineEdgeMaps(std::vector<cv::Mat>& edgeMaps, cv::Mat& output);

void processCannySingleImage(const cv::Mat& inputImage, cv::Mat& outputEdges) {
    int width = inputImage.cols;
    int height = inputImage.rows;

    // Copy the Gaussian kernel to constant memory
    const float h_kernel[5][5] = {
        {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
        {4 / 273.0f,16 / 273.0f,26 / 273.0f,16 / 273.0f, 4 / 273.0f},
        {7 / 273.0f,26 / 273.0f,41 / 273.0f,26 / 273.0f, 7 / 273.0f},
        {4 / 273.0f,16 / 273.0f,26 / 273.0f,16 / 273.0f, 4 / 273.0f},
        {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}
    };

    // Start timing for kernel copy
    auto kernelCopyStart = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, sizeof(h_kernel)));
    auto kernelCopyEnd = std::chrono::high_resolution_clock::now();
    auto kernelCopyDuration = std::chrono::duration_cast<std::chrono::milliseconds>(kernelCopyEnd - kernelCopyStart).count();
    std::cout << "Kernel copy to constant memory time: " << kernelCopyDuration << " ms" << std::endl;

    // Allocate device memory
    float *d_image, *d_edges;
    auto deviceAllocStart = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_edges, width * height * sizeof(float)));
    auto deviceAllocEnd = std::chrono::high_resolution_clock::now();
    auto deviceAllocDuration = std::chrono::duration_cast<std::chrono::milliseconds>(deviceAllocEnd - deviceAllocStart).count();
    std::cout << "Device memory allocation time: " << deviceAllocDuration << " ms" << std::endl;

    // Copy image to device
    float* h_image = (float*)malloc(width * height * sizeof(float));
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            h_image[i * width + j] = inputImage.at<uchar>(i, j) / 255.0f;
        }
    }
    auto copyToDeviceStart = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpy(d_image, h_image, width * height * sizeof(float), hipMemcpyHostToDevice));
    auto copyToDeviceEnd = std::chrono::high_resolution_clock::now();
    auto copyToDeviceDuration = std::chrono::duration_cast<std::chrono::milliseconds>(copyToDeviceEnd - copyToDeviceStart).count();
    std::cout << "Image copy to device time: " << copyToDeviceDuration << " ms" << std::endl;

    // Process single scale (includes Gaussian smoothing)
    auto processingStart = std::chrono::high_resolution_clock::now();
    processSingleScale(d_image, d_edges, width, height);
    auto processingEnd = std::chrono::high_resolution_clock::now();
    auto processingDuration = std::chrono::duration_cast<std::chrono::microseconds>(processingEnd - processingStart).count();
    std::cout << "Single-scale processing time: " << processingDuration << " µs" << std::endl;

    // Copy result back to host
    float* h_edges = (float*)malloc(width * height * sizeof(float));
    auto copyToHostStart = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpy(h_edges, d_edges, width * height * sizeof(float), hipMemcpyDeviceToHost));
    auto copyToHostEnd = std::chrono::high_resolution_clock::now();
    auto copyToHostDuration = std::chrono::duration_cast<std::chrono::milliseconds>(copyToHostEnd - copyToHostStart).count();
    std::cout << "Result copy to host time: " << copyToHostDuration << " ms" << std::endl;

    // Convert result to OpenCV format
    auto edgeConversionStart = std::chrono::high_resolution_clock::now();
    cv::Mat edgeMap(height, width, CV_32F, h_edges);
    edgeMap.convertTo(outputEdges, CV_8U, 255.0);
    auto edgeConversionEnd = std::chrono::high_resolution_clock::now();
    auto edgeConversionDuration = std::chrono::duration_cast<std::chrono::milliseconds>(edgeConversionEnd - edgeConversionStart).count();
    std::cout << "Edge map conversion time: " << edgeConversionDuration << " ms" << std::endl;

    // Free memory
    auto freeMemoryStart = std::chrono::high_resolution_clock::now();
    free(h_image);
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_edges));
    auto freeMemoryEnd = std::chrono::high_resolution_clock::now();
    auto freeMemoryDuration = std::chrono::duration_cast<std::chrono::milliseconds>(freeMemoryEnd - freeMemoryStart).count();
    std::cout << "Memory deallocation time: " << freeMemoryDuration << " ms" << std::endl;
}



// Multi-Scale Canny Implementation
void multiScaleCanny(const cv::Mat& inputImage, std::vector<float> scales, cv::Mat& outputEdges) {
    int originalWidth = inputImage.cols;
    int originalHeight = inputImage.rows;

    std::vector<cv::Mat> edgeMaps;

    // For total time of multiScaleCanny
    auto totalStart = std::chrono::high_resolution_clock::now();
    const float h_kernel[5][5] = {
        {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
        {4 / 273.0f,16 / 273.0f,26 / 273.0f,16 / 273.0f, 4 / 273.0f},
        {7 / 273.0f,26 / 273.0f,41 / 273.0f,26 / 273.0f, 7 / 273.0f},
        {4 / 273.0f,16 / 273.0f,26 / 273.0f,16 / 273.0f, 4 / 273.0f},
        {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}
    };

    // Copy the Gaussian kernel to constant memory
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, sizeof(h_kernel)));

    for (float scale : scales) {
        // Start timer for this scale
        auto scaleStart = std::chrono::high_resolution_clock::now();

        // Resize the image for the current scale
        auto resizeStart = std::chrono::high_resolution_clock::now();
        int scaledWidth = static_cast<int>(originalWidth * scale);
        int scaledHeight = static_cast<int>(originalHeight * scale);
        cv::Mat resizedImage;
        cv::resize(inputImage, resizedImage, cv::Size(scaledWidth, scaledHeight), 0, 0, cv::INTER_LINEAR);
        auto resizeEnd = std::chrono::high_resolution_clock::now();
        auto resizeDuration = std::chrono::duration_cast<std::chrono::milliseconds>(resizeEnd - resizeStart).count();

        // Copy the resized image to device memory
        auto hostAllocStart = std::chrono::high_resolution_clock::now();
        float* h_resizedImage = (float*)malloc(scaledWidth * scaledHeight * sizeof(float));
        for (int i = 0; i < resizedImage.rows; ++i) {
            for (int j = 0; j < resizedImage.cols; ++j) {
                h_resizedImage[i * scaledWidth + j] = resizedImage.at<uchar>(i, j) / 255.0f;
            }
        }
        auto hostAllocEnd = std::chrono::high_resolution_clock::now();
        auto hostAllocDuration = std::chrono::duration_cast<std::chrono::milliseconds>(hostAllocEnd - hostAllocStart).count();

        // Device memory allocation
        auto deviceAllocStart = std::chrono::high_resolution_clock::now();
        float *d_image, *d_edges;
        CUDA_CHECK(hipMalloc(&d_image, scaledWidth * scaledHeight * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_edges, scaledWidth * scaledHeight * sizeof(float)));
        auto deviceAllocEnd = std::chrono::high_resolution_clock::now();
        auto deviceAllocDuration = std::chrono::duration_cast<std::chrono::milliseconds>(deviceAllocEnd - deviceAllocStart).count();

        // Copy to device
        auto copyToDeviceStart = std::chrono::high_resolution_clock::now();
        CUDA_CHECK(hipMemcpy(d_image, h_resizedImage, scaledWidth * scaledHeight * sizeof(float), hipMemcpyHostToDevice));
        auto copyToDeviceEnd = std::chrono::high_resolution_clock::now();
        auto copyToDeviceDuration = std::chrono::duration_cast<std::chrono::milliseconds>(copyToDeviceEnd - copyToDeviceStart).count();
        float transferTimeSec = copyToDeviceDuration / 1000.0f;
        float bandwidth = (scaledWidth * scaledHeight * sizeof(float)) / (1024.0 * 1024.0 * 1024.0) / transferTimeSec;
        std::cout << "  Copy to device bandwidth: " << bandwidth << " GB/s" << std::endl;

        // Process single scale
        auto processStart = std::chrono::high_resolution_clock::now();
        processSingleScale(d_image, d_edges, scaledWidth, scaledHeight);
        auto processEnd = std::chrono::high_resolution_clock::now();
        auto processDuration = std::chrono::duration_cast<std::chrono::milliseconds>(processEnd - processStart).count();

        // Copy the edge map back to host memory
        auto copyToHostStart = std::chrono::high_resolution_clock::now();
        float* h_edges = (float*)malloc(scaledWidth * scaledHeight * sizeof(float));
        CUDA_CHECK(hipMemcpy(h_edges, d_edges, scaledWidth * scaledHeight * sizeof(float), hipMemcpyDeviceToHost));
        auto copyToHostEnd = std::chrono::high_resolution_clock::now();
        auto copyToHostDuration = std::chrono::duration_cast<std::chrono::milliseconds>(copyToHostEnd - copyToHostStart).count();
        transferTimeSec = copyToHostDuration / 1000.0f;
        bandwidth = (scaledWidth * scaledHeight * sizeof(float)) / (1024.0 * 1024.0 * 1024.0) / transferTimeSec;
        std::cout << "  Copy to host bandwidth: " << bandwidth << " GB/s" << std::endl;

        // Resize edge map back to original size
        auto edgeResizeStart = std::chrono::high_resolution_clock::now();
        cv::Mat edgeMap(scaledHeight, scaledWidth, CV_32F, h_edges);
        cv::resize(edgeMap, edgeMap, cv::Size(originalWidth, originalHeight), 0, 0, cv::INTER_LINEAR);
        edgeMaps.push_back(edgeMap);
        auto edgeResizeEnd = std::chrono::high_resolution_clock::now();
        auto edgeResizeDuration = std::chrono::duration_cast<std::chrono::milliseconds>(edgeResizeEnd - edgeResizeStart).count();

        // Free memory
        auto freeStart = std::chrono::high_resolution_clock::now();
        free(h_resizedImage);
        // Do not free h_edges here; it is managed by cv::Mat
        CUDA_CHECK(hipFree(d_image));
        CUDA_CHECK(hipFree(d_edges));
        auto freeEnd = std::chrono::high_resolution_clock::now();
        auto freeDuration = std::chrono::duration_cast<std::chrono::milliseconds>(freeEnd - freeStart).count();

        auto scaleEnd = std::chrono::high_resolution_clock::now();
        auto scaleDuration = std::chrono::duration_cast<std::chrono::milliseconds>(scaleEnd - scaleStart).count();

        // Print timings for this scale
        std::cout << "Scale: " << scale << " - Time: " << scaleDuration << " ms" << std::endl;
        std::cout << "  Resize image: " << resizeDuration << " ms" << std::endl;
        std::cout << "  Host allocation and copy: " << hostAllocDuration << " ms" << std::endl;
        std::cout << "  Device memory allocation: " << deviceAllocDuration << " ms" << std::endl;
        std::cout << "  Copy to device: " << copyToDeviceDuration << " ms" << std::endl;
        std::cout << "  Processing (kernels): " << processDuration << " ms" << std::endl;
        std::cout << "  Copy to host: " << copyToHostDuration << " ms" << std::endl;
        std::cout << "  Resize edge map: " << edgeResizeDuration << " ms" << std::endl;
        std::cout << "  Free memory: " << freeDuration << " ms" << std::endl;
    }

    // Combine edge maps from all scales
    auto combineStart = std::chrono::high_resolution_clock::now();
    combineEdgeMaps(edgeMaps, outputEdges);
    auto combineEnd = std::chrono::high_resolution_clock::now();
    auto combineDuration = std::chrono::duration_cast<std::chrono::milliseconds>(combineEnd - combineStart).count();

    auto totalEnd = std::chrono::high_resolution_clock::now();
    auto totalDuration = std::chrono::duration_cast<std::chrono::milliseconds>(totalEnd - totalStart).count();

    std::cout << "Total multiScaleCanny time: " << totalDuration << " ms" << std::endl;
    std::cout << "  Combine edge maps: " << combineDuration << " ms" << std::endl;
}

void processSingleScale(float* d_image, float* d_output, int width, int height) {
    float *d_blurred, *d_gradient, *d_direction;
    CUDA_CHECK(hipMalloc(&d_blurred, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gradient, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_direction, width * height * sizeof(float)));

    // Gaussian Blur
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Create CUDA events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    // Gaussian Blur
    int minGridSize, blockSize1;
    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize1, GaussianBlurKernel, 0, 0));
    std::cout << "    GaussianBlurKernel occupancy: " << blockSize1 
            << " threads per block (min grid size: " << minGridSize << ")" << std::endl;
    CUDA_CHECK(hipEventRecord(start));
    GaussianBlurKernel<<<gridSize, blockSize>>>(d_image, d_blurred, width, height);
    hipFuncAttributes attributes;
    CUDA_CHECK(hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(GaussianBlurKernel)));
    std::cout << "    GaussianBlurKernel shared memory usage: " 
            << attributes.sharedSizeBytes << " bytes" << std::endl;
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "    GaussianBlurKernel time: " << milliseconds << " ms" << std::endl;

    // Gradient Computation
    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize1, SobelKernel, 0, 0));
    std::cout << "    SobelKernel occupancy: " << blockSize1 
            << " threads per block (min grid size: " << minGridSize << ")" << std::endl;
    CUDA_CHECK(hipEventRecord(start));
    SobelKernel<<<gridSize, blockSize>>>(d_blurred, d_gradient, d_direction, width, height);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "    SobelKernel time: " << milliseconds << " ms" << std::endl;
    CUDA_CHECK(hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(SobelKernel)));
    std::cout << "    SobelKernel shared memory usage: " 
          << attributes.sharedSizeBytes << " bytes" << std::endl;
    // Non-Maximum Suppression
    CUDA_CHECK(hipEventRecord(start));
    NonMaxSuppressionKernel<<<gridSize, blockSize>>>(d_gradient, d_direction, d_output, width, height);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "    NonMaxSuppressionKernel time: " << milliseconds << " ms" << std::endl;

    // Double Thresholding
    CUDA_CHECK(hipEventRecord(start));
    DoubleThresholdKernel<<<gridSize, blockSize>>>(d_output, 0.2f, 0.1f, width, height);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "    DoubleThresholdKernel time: " << milliseconds << " ms" << std::endl;

    // Edge Tracking by Hysteresis
    CUDA_CHECK(hipEventRecord(start));
    HysteresisKernel<<<gridSize, blockSize>>>(d_output, width, height);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "    HysteresisKernel time: " << milliseconds << " ms" << std::endl;

    // Free intermediate buffers
    CUDA_CHECK(hipFree(d_blurred));
    CUDA_CHECK(hipFree(d_gradient));
    CUDA_CHECK(hipFree(d_direction));

    // Destroy CUDA events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void combineEdgeMaps(std::vector<cv::Mat>& edgeMaps, cv::Mat& output) {
    output = cv::Mat::zeros(edgeMaps[0].size(), CV_32F);

    for (const cv::Mat& edgeMap : edgeMaps) {
        cv::max(output, edgeMap, output);
    }

    // Convert back to 8-bit for display
    output.convertTo(output, CV_8U, 255.0);
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <image_file> <mode> [scales]" << std::endl;
        std::cerr << "  <mode>: 'single' or 'multi'" << std::endl;
        std::cerr << "  [scales]: Comma-separated list of scales for 'multi' mode (e.g., 0.5,1.0,2.0)" << std::endl;
        return -1;
    }
    hipFree(0);
    const char* inputImageFile = argv[1];
    std::string mode = argv[2];

    // Load input image
    cv::Mat inputImage = cv::imread(inputImageFile, cv::IMREAD_GRAYSCALE);
    if (inputImage.empty()) {
        std::cerr << "Error: Could not load image." << std::endl;
        return -1;
    }

    cv::Mat outputEdges;

    if (mode == "single") {
        // Perform Canny edge detection without multi-scale Gaussian
        std::cout << "Running single-scale Canny edge detection..." << std::endl;
        processCannySingleImage(inputImage, outputEdges);
    } else if (mode == "multi") {
        // Parse scales from command line (default: {0.5, 1.0, 2.0})
        std::vector<float> scales = {0.5f, 1.0f, 2.0f};
        if (argc == 4) {
            scales.clear();
            std::string scaleArg = argv[3];
            size_t pos = 0;
            while ((pos = scaleArg.find(',')) != std::string::npos) {
                scales.push_back(std::stof(scaleArg.substr(0, pos)));
                scaleArg.erase(0, pos + 1);
            }
            scales.push_back(std::stof(scaleArg)); // Add the last scale
        }

        std::cout << "Running multi-scale Canny edge detection with scales: ";
        for (float scale : scales) std::cout << scale << " ";
        std::cout << std::endl;

        multiScaleCanny(inputImage, scales, outputEdges);
    } else {
        std::cerr << "Error: Unknown mode '" << mode << "'. Use 'single' or 'multi'." << std::endl;
        return -1;
    }

    // Save and display the output
    cv::imwrite("canny_edges.png", outputEdges);
    // cv::imshow("Canny Edges", outputEdges);
    cv::waitKey(0);

    return 0;
}